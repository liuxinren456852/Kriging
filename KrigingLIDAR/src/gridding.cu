#include "hip/hip_runtime.h"
// example1.cpp : Defines the entry point for the console application.
//

//#include <stdafx.h>

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <time.h>
#include <float.h>
#include <math.h>

#include "cuPrintf.h"

void findMinMax (double x, double y);

//Maximum & Minimum variables
float minX=0.0, minY=0.0, maxX=0.0, maxY=0.0;

__device__ int Doolittle_LU_Decomposition_with_Pivoting(double *A, int pivot[], int n)
{
	int i, j, k;
	double *p_k, *p_row, *p_col;
	double max;

	//cuPrintf("nilai n %d\n",n);
	int id=0;
	for(i=0;i<n;i++){
		for(j=0;j<n;j++){
			//cuPrintf("%f; ",*(A+id));
			id++;
		}
		//cuPrintf("\n");
	}

	//         For each row and column, k = 0, ..., n-1,

	for (k = 0, p_k = A; k < n; p_k += n, k++) {

		//            find the pivot row

		pivot[k] = k;
		//cuPrintf("*(p_k+k) = %f\n",*(p_k+k));
		max = fabs( *(p_k + k) );
		for (j = k + 1, p_row = p_k + n; j < n; j++, p_row += n) {
			if ( max < fabs(*(p_row + k) )){
				max = fabs(*(p_row + k));//*(p_row + k) =  *A
				pivot[k] = j;
				p_col = p_row;
			}
		}

		//     and if the pivot row differs from the current row, then
		//     interchange the two rows.

		if (pivot[k] != k)
			for (j = 0; j < n; j++) {
				max = *(p_k + j);
				*(p_k + j) = *(p_col + j);
				*(p_col + j) = max;
			}

		//                and if the matrix is singular, return error


		if ( *(p_k + k) == 0.0 ) return -1;

		//      otherwise find the lower triangular matrix elements for column k.

		for (i = k+1, p_row = p_k + n; i < n; p_row += n, i++) {
			*(p_row + k) /= *(p_k + k);
		}

		//            update remaining matrix

		for (i = k+1, p_row = p_k + n; i < n; p_row += n, i++)
			for (j = k+1; j < n; j++)
				*(p_row + j) -= *(p_row + k) * *(p_k + j);

	}

	return 0;
}


__device__ int Doolittle_LU_with_Pivoting_Solve(double *A, double B[], int pivot[],
		double x[], int n)
{
	int i, k;
	double *p_k;
	double dum;

	//         Solve the linear equation Lx = B for x, where L is a lower
	//         triangular matrix with an implied 1 along the diagonal.

	for (k = 0, p_k = A; k < n; p_k += n, k++) {
		if (pivot[k] != k) {dum = B[k]; B[k] = B[pivot[k]]; B[pivot[k]] = dum; }
		x[k] = B[k];
		for (i = 0; i < k; i++) x[k] -= x[i] * *(p_k + i);
	}

	//         Solve the linear equation Ux = y, where y is the solution
	//         obtained above of Lx = B and U is an upper triangular matrix.

	for (k = n-1, p_k = A + n*(n-1); k >= 0; k--, p_k -= n) {
		if (pivot[k] != k) {dum = B[k]; B[k] = B[pivot[k]]; B[pivot[k]] = dum; }
		for (i = k + 1; i < n; i++) x[k] -= x[i] * *(p_k + i);
		if (*(p_k + k) == 0.0) return -1;
		x[k] /= *(p_k + k);
	}

	return 0;
}

__device__ double sphericalModel (double range, double sill, double h){
	return sill*powf( (3*h/(2*range)) - 1/2*(h/range) , 3);
	//	return 10*(1-exp(-h/3.33));
}

__device__ double leastsquaredSumFunc (double *semivar, double* b0, double* h, int nrbins){
	double sum=0.0;
	int i;
	/*for (i=0 ; i<nrbins ; i++){
		printf("semivar: %lf, distBins: %lf\n", semivar[i], h[i]);
	}
	printf("b: %lf,%lf,  nrbins: %d\n", b0[0], b0[1], nrbins);*/
	for (i=0 ; i<nrbins ; i++){
		if (h[i] < b0[0]){
			sum += powf( ( b0[1]*(3*h[i]/(2*b0[0])- 1/2*(h[i]/b0[0]))*(3*h[i]/(2*b0[0])- 1/2*(h[i]/b0[0]))*(3*h[i]/(2*b0[0])- 1/2*(h[i]/b0[0]))-semivar[i]) ,2);
			//sum+=pow(sphericalModel(b0[0], b0[1], h[i])-semivar[i], 2);
		}else{
			sum+=pow(b0[1]-semivar[i], 2);
		}

	}
	return sum;
}

__global__ void on_core_process(int GPUIndex, double* ptrIsFinished, double *x_device, double *y_device, double *z_device, double minimumX, double minimumY, int numInputData)
{

	//Specify anchor coordinates.
	//We calculate distance from this point.
	//This point will be different from each GPU thread
	//We assume that grid process is per 1 meter

	int gridID = blockIdx.x+(blockIdx.y*gridDim.x);
	//retrieve thread ID
	int threadID = ((threadIdx.y)*blockDim.x+(threadIdx.x))+gridID*gridDim.x*gridDim.y;
	threadID = threadID+500000*GPUIndex;
	double y_node = (double) (1*floor((float)threadID/1000));
	//shift load jobs into several devices
//	threadID = threadID + GPUIndex*500000; // 500000 is number of grid divide by number of devices
	double x_node = (double) (threadID-1000*(floor((double)threadID/1000)));
//	x_node = x_node+(double)(500*GPUIndex);


	//debug version
	//	double y_node = 1*floor((float)threadID/50);
	//	double x_node = threadID-50*(floor((double)threadID/50));

	//do each thread process here
	int numberOfGrids = 1000*1000;
	double weightsum;
	int idx_onRange;

	if (threadID<numberOfGrids) {
		//	if (x_node==(65-minimumX) && y_node==(137-minimumY)) {
		//cuPrintf("Horee\n");
		//Calculate distance from each LIDAR input and save it into index if the distance still on the range
		// in here we assume that we have only 1000 closest nodes
		float x_closestNodesIndex[100];
		float y_closestNodesIndex[100];
		float z_closestNodesIndex[100];

		int gridRadius = 3;
		idx_onRange=0;
		for(int i=0;i<numInputData;i++){ //TODO to big numInputData make this program crash
			//save the value if distance is still on range
			if(( abs((x_device[i]-(x_node+minimumX)))<gridRadius)&&(abs((y_device[i]-(y_node+minimumY)))<gridRadius)){
				x_closestNodesIndex[idx_onRange] = (float)x_device[i]; //change this with pointer to enhance performance
				y_closestNodesIndex[idx_onRange] = (float)y_device[i]; //change this with pointer to enhance performance
				z_closestNodesIndex[idx_onRange] = (float)z_device[i];
				if(idx_onRange<100){ //TODO solve this function! why more tha
					idx_onRange++;
				}
			}
		}
		//cuPrintf("idx_onRange %d \n",idx_onRange);
		//cuPrintf("numinput %d \n",numInputData);
		//TODO find min & max value from closestNodesIndex variable (both X & Y)
		double min_x=999999;
		double max_x=0;
		double min_y=9999999;
		double max_y=0;
		for(int i=0;i<idx_onRange;i++){
			if(x_closestNodesIndex[i]<min_x)
				min_x = x_closestNodesIndex[i];
			if(y_closestNodesIndex[i]<min_y)
				min_y = y_closestNodesIndex[i];
			if(y_closestNodesIndex[i]>max_y)
				max_y = y_closestNodesIndex[i];
			if(x_closestNodesIndex[i]>max_x)
				max_x = x_closestNodesIndex[i];
			i++;
		}

		//Calculate Semivariogram
		int nrbins = 50; //this is a parameter
		int rows = idx_onRange;
		double occIdxdistBins[50]; //parameter : nrbins
		memset(occIdxdistBins,0,sizeof(occIdxdistBins));
		double sumSqurZ[50]; //parameter : nrbins
		memset(sumSqurZ,0,sizeof(sumSqurZ));
		double distance[50]; //parameter : nrbins
		memset(distance,0,sizeof(distance));
		float ptopDistance[100][100];
		memset(ptopDistance,0,sizeof(ptopDistance));


		float predist = sqrtf((float)(powf(max_x-min_x,2)+powf(max_y-min_y,2)));
		float maxdist = predist/2;
		float delta = maxdist/2;

		//cuPrintf("masdist: %f \n",maxdist);
		for(int i=0;i<rows;i++){
			for(int j=i+1; j<rows;j++){
				ptopDistance[i][j]= sqrt((float)(powf((float)(x_closestNodesIndex[i]-x_closestNodesIndex[j]),2.0)
						+powf((float)(y_closestNodesIndex[i]-y_closestNodesIndex[j]),2.0)));

				if(ptopDistance[i][j]<maxdist){
					int idx = (int) floorf(ptopDistance[i][j]/delta);
					occIdxdistBins[idx] = occIdxdistBins[idx]+1;
					double squrZ =powf(z_closestNodesIndex[i]-z_closestNodesIndex[j],2);
					//cuPrintf("d[%d,%d]:%f\n",i,j,z_closestNodesIndex[i]);
					//cuPrintf("squrZ:%f\n", squrZ);
					sumSqurZ[idx]=sumSqurZ[idx]+squrZ;
					//cuPrintf("ptopDistance[%d][%d]:%f\n",i,j,ptopDistance[i][j]);
				}
			}
		}

		for(int i=0;i<nrbins; i++){
			if(occIdxdistBins[i] == 0.0){
				sumSqurZ[i] = 0.0;
			}else{
				sumSqurZ[i] = sumSqurZ[i]/(2*occIdxdistBins[i]);
			}
			//cuPrintf("sumSqurZ[%d] : %f \n",i,sumSqurZ[i]);
		}

		distance[0]=delta/2;
		for(int i=0; i<nrbins;i++){
			distance[i]=distance[i-1]+delta;
			//			//cuPrintf("distance[%d]:%f\n",i,distance[i]);
		}

		/** SEmivariogram process is completed here*/

		/**
		 * Starting Fitting
		 */
		double maxVario, minLs[3] = {DBL_MAX,0,0}, tmpLS[3]; // For fit
		double lagbin, semibin;
		double b0[3] = {0};

		//find maximum variogram value
		double max = 0.0;
		for(int i=0;i<nrbins;i++){
			if(sumSqurZ[i]>max)
				max = sumSqurZ[i];
		}
		maxVario = max;

		int bins = 10;
		//b0[0] = range
		//b0[1] = sill
		b0[0] = distance[nrbins-1] * 2 /3;
		b0[1] = maxVario;
		lagbin = b0[0] / bins;
		semibin = b0[1]/ bins;
		int i,j;
		for (i=0 ; i<bins ; i++){
			for (j=0 ; j<bins ; j++){
				b0[0] -= lagbin;
				tmpLS[0] = leastsquaredSumFunc(sumSqurZ, b0, distance, nrbins);
				tmpLS[1]= b0[0];
				tmpLS[2]= b0[1];
				if(minLs[0] > tmpLS[0]) {
					minLs[0] = tmpLS[0];
					minLs[1] = b0[0];
					minLs[2] = b0[1];
				}

			}
			b0 [0] = distance[nrbins-1] * 2 / 3;
			b0 [1] -= semibin;
		}

		if (minLs[1] < 0.0001){
			minLs[1] = distance[0];
		}

		double range = minLs[1];
		double sill = minLs[2];
		//		range = 200;//Hard code
		//		sill = 150;
		//cuPrintf("range:%f\n",range);
		//cuPrintf("sill:%f\n",sill);

		/*
		 * mulai dari sini yang belom FIX...huaaa...huaaa...
		 *
		 *
		//prediction(struct Distnode *Head_distfromGrid,
		                  int numofNearestPoint,
		                  double range, double sill,
		                  int numofPoints,  --> idx_onRange
		                  int gridX, int gridY,
		                  double coord_gridX, double coord_gridY,
		                  double** distance) ptopDistance


		 */
		int N = idx_onRange+1; // we need one more column  & Row to fill 0 and 1 values (Ordinary Kriging)
		double predict=0;

		//TODO calculate distance achor node with
		int counter_idxClosestRange =0;
		while (counter_idxClosestRange < idx_onRange){
			ptopDistance[counter_idxClosestRange][N-1] = sqrt((float)(powf((float)(x_closestNodesIndex[counter_idxClosestRange]-(x_node+minimumX)),2.0)
					+powf((float)(y_closestNodesIndex[counter_idxClosestRange]-(y_node+minimumY)),2.0)));
			//cuPrintf("ptopDistance[%d][%d]:%f\n",counter_idxClosestRange,N-1,ptopDistance[counter_idxClosestRange][N-1]);
			counter_idxClosestRange++;
		}

		//		for(int it=0;it<counter_idxClosestRange+1;it++){
		//			//cuPrintf("ptopDistance: %2.2f \n",ptopDistance[it]);
		//		}

		double a[10201]; //101x101
		double *a_ptr = a;
		double rptr[101][101];//double **rptr;
		//		a = (double*) rptr;

		double b[101];
		int pivot[101];
		double sol[101];
		memset(rptr,0,sizeof(rptr));
		memset(b,0,sizeof(b));
		memset(pivot,0,sizeof(pivot));
		memset(sol,0,sizeof(sol));

		//----------- Applied Gamma Function for Ordinary Kriging
		for(i=0;i<N;i++){
			for(j=i;j<N;j++){
				if(j==N-1) {
					rptr[i][j] = -1.0;
					//cuPrintf("%f < %f kah?? \n",ptopDistance [i][j],range );
					if (ptopDistance [i][j] < range){
						b[i] = sphericalModel(range, sill, ptopDistance[i][j]);
					}
					else{
						b[i] = sill;
					}
					//					//cuPrintf("b[%d]:%f\n",i,b[i]);
				}else{
					if (ptopDistance [i][j] < range){
						rptr[i][j] = sphericalModel(range, sill, ptopDistance[i][j]);
					}
					else{
						rptr[i][j] = sill;
					}
					//					//cuPrintf("rptr[%d]:%f\n",i,rptr[i]);
					rptr[j][i] = rptr[i][j];
					rptr[N-1][j]=1.0;
				}
			}
		}
		rptr[N-1][N-1] = 0.0;
		b[N-1] = 1.0;

		//TODO change this with more efficient way
		for(i=0;i<N;i++){
			for(j=0;j<N;j++){
				//cuPrintf("rptr[%d][%d]:%f\n",i,j,rptr[i][j]);
				*a_ptr = rptr[i][j];
				a_ptr++;
			}
		}

		//		for(i=0;i<N;i++){
		//cuPrintf("b[%d]:%f\n",i,b[i]);
		//		}

		int err = Doolittle_LU_Decomposition_with_Pivoting(a, pivot,  N);
		//cuPrintf("N=%d \n",N);
		//cuPrintf("err: %d\n",err);
		//printf ("after decom\n");
		if (err < 0) {//cuPrintf("matrix is Singular\n");
			//printf(" Matrix A is singular\n");
		}
		else {
			err = Doolittle_LU_with_Pivoting_Solve(a, b, pivot, sol, N);

		}

		weightsum=0.0;
		predict = 0.0;
		//calculate predict value in grid or radius or nearest points
		for (i=0 ; i<N-1 ; i++){
			//cuPrintf("sol[%d]:%f\n",i,sol[i]);
			predict += z_closestNodesIndex[i] * sol[i];
			weightsum += sol[i];
		}

		//cuPrintf("ID[%d];weightsum: %f\n",threadID,weightsum);
		//cuPrintf("ID[%d];predict %f\n",threadID,predict);
		//		//cuPrintf("sill[%d]: %2.2f\n", threadID, sill);
		//		//cuPrintf("range[%d]: %2.2f\n", threadID, range);


	}
	int temp = threadID;
	threadID = threadID-500000*GPUIndex;
	//assign flag complete notification
	ptrIsFinished[threadID]=temp;
}

// Kernel that executes on the CUDA device
__global__ void on_core_process2(int GPUIndex, double* ptrIsFinished, double *x_device, double *y_device, double *z_device, double minimumX, double minimumY, int numInputData)
{

	//Specify anchor coordinates.
	//We calculate distance from this point.
	//This point will be different from each GPU thread
	//We assume that grid process is per 1 meter

	int gridID = blockIdx.x+(blockIdx.y*gridDim.x);
	//retrieve thread ID
	int threadID = ((threadIdx.y)*blockDim.x+(threadIdx.x))+gridID*gridDim.x*gridDim.y;
	threadID = threadID+500000*GPUIndex;
	double y_node = (double) (1*floor((float)threadID/1000));
	//shift load jobs into several devices
//	threadID = threadID + GPUIndex*500000; // 500000 is number of grid divide by number of devices
	double x_node = (double) (threadID-1000*(floor((double)threadID/1000)));
//	x_node = x_node+(double)(500*GPUIndex);


	//debug version
	//	double y_node = 1*floor((float)threadID/50);
	//	double x_node = threadID-50*(floor((double)threadID/50));

	//do each thread process here
	int numberOfGrids = 1000*1000;
	double weightsum;
	int idx_onRange;

	if (threadID<numberOfGrids) {
		//	if (x_node==(65-minimumX) && y_node==(137-minimumY)) {
		//cuPrintf("Horee\n");
		//Calculate distance from each LIDAR input and save it into index if the distance still on the range
		// in here we assume that we have only 1000 closest nodes
		float x_closestNodesIndex[100];
		float y_closestNodesIndex[100];
		float z_closestNodesIndex[100];

		int gridRadius = 3;
		idx_onRange=0;
		for(int i=0;i<numInputData;i++){ //TODO to big numInputData make this program crash
			//save the value if distance is still on range
			if(( abs((x_device[i]-(x_node+minimumX)))<gridRadius)&&(abs((y_device[i]-(y_node+minimumY)))<gridRadius)){
				x_closestNodesIndex[idx_onRange] = (float)x_device[i]; //change this with pointer to enhance performance
				y_closestNodesIndex[idx_onRange] = (float)y_device[i]; //change this with pointer to enhance performance
				z_closestNodesIndex[idx_onRange] = (float)z_device[i];
				if(idx_onRange<100){ //TODO solve this function! why more tha
					idx_onRange++;
				}
			}
		}
		//cuPrintf("idx_onRange %d \n",idx_onRange);
		//cuPrintf("numinput %d \n",numInputData);
		//TODO find min & max value from closestNodesIndex variable (both X & Y)
		double min_x=999999;
		double max_x=0;
		double min_y=9999999;
		double max_y=0;
		for(int i=0;i<idx_onRange;i++){
			if(x_closestNodesIndex[i]<min_x)
				min_x = x_closestNodesIndex[i];
			if(y_closestNodesIndex[i]<min_y)
				min_y = y_closestNodesIndex[i];
			if(y_closestNodesIndex[i]>max_y)
				max_y = y_closestNodesIndex[i];
			if(x_closestNodesIndex[i]>max_x)
				max_x = x_closestNodesIndex[i];
			i++;
		}

		//Calculate Semivariogram
		int nrbins = 50; //this is a parameter
		int rows = idx_onRange;
		double occIdxdistBins[50]; //parameter : nrbins
		memset(occIdxdistBins,0,sizeof(occIdxdistBins));
		double sumSqurZ[50]; //parameter : nrbins
		memset(sumSqurZ,0,sizeof(sumSqurZ));
		double distance[50]; //parameter : nrbins
		memset(distance,0,sizeof(distance));
		float ptopDistance[100][100];
		memset(ptopDistance,0,sizeof(ptopDistance));


		float predist = sqrtf((float)(powf(max_x-min_x,2)+powf(max_y-min_y,2)));
		float maxdist = predist/2;
		float delta = maxdist/2;

		//cuPrintf("masdist: %f \n",maxdist);
		for(int i=0;i<rows;i++){
			for(int j=i+1; j<rows;j++){
				ptopDistance[i][j]= sqrt((float)(powf((float)(x_closestNodesIndex[i]-x_closestNodesIndex[j]),2.0)
						+powf((float)(y_closestNodesIndex[i]-y_closestNodesIndex[j]),2.0)));

				if(ptopDistance[i][j]<maxdist){
					int idx = (int) floorf(ptopDistance[i][j]/delta);
					occIdxdistBins[idx] = occIdxdistBins[idx]+1;
					double squrZ =powf(z_closestNodesIndex[i]-z_closestNodesIndex[j],2);
					//cuPrintf("d[%d,%d]:%f\n",i,j,z_closestNodesIndex[i]);
					//cuPrintf("squrZ:%f\n", squrZ);
					sumSqurZ[idx]=sumSqurZ[idx]+squrZ;
					//cuPrintf("ptopDistance[%d][%d]:%f\n",i,j,ptopDistance[i][j]);
				}
			}
		}

		for(int i=0;i<nrbins; i++){
			if(occIdxdistBins[i] == 0.0){
				sumSqurZ[i] = 0.0;
			}else{
				sumSqurZ[i] = sumSqurZ[i]/(2*occIdxdistBins[i]);
			}
			//cuPrintf("sumSqurZ[%d] : %f \n",i,sumSqurZ[i]);
		}

		distance[0]=delta/2;
		for(int i=0; i<nrbins;i++){
			distance[i]=distance[i-1]+delta;
			//			//cuPrintf("distance[%d]:%f\n",i,distance[i]);
		}

		/** SEmivariogram process is completed here*/

		/**
		 * Starting Fitting
		 */
		double maxVario, minLs[3] = {DBL_MAX,0,0}, tmpLS[3]; // For fit
		double lagbin, semibin;
		double b0[3] = {0};

		//find maximum variogram value
		double max = 0.0;
		for(int i=0;i<nrbins;i++){
			if(sumSqurZ[i]>max)
				max = sumSqurZ[i];
		}
		maxVario = max;

		int bins = 10;
		//b0[0] = range
		//b0[1] = sill
		b0[0] = distance[nrbins-1] * 2 /3;
		b0[1] = maxVario;
		lagbin = b0[0] / bins;
		semibin = b0[1]/ bins;
		int i,j;
		for (i=0 ; i<bins ; i++){
			for (j=0 ; j<bins ; j++){
				b0[0] -= lagbin;
				tmpLS[0] = leastsquaredSumFunc(sumSqurZ, b0, distance, nrbins);
				tmpLS[1]= b0[0];
				tmpLS[2]= b0[1];
				if(minLs[0] > tmpLS[0]) {
					minLs[0] = tmpLS[0];
					minLs[1] = b0[0];
					minLs[2] = b0[1];
				}

			}
			b0 [0] = distance[nrbins-1] * 2 / 3;
			b0 [1] -= semibin;
		}

		if (minLs[1] < 0.0001){
			minLs[1] = distance[0];
		}

		double range = minLs[1];
		double sill = minLs[2];
		//		range = 200;//Hard code
		//		sill = 150;
		//cuPrintf("range:%f\n",range);
		//cuPrintf("sill:%f\n",sill);

		/*
		 * mulai dari sini yang belom FIX...huaaa...huaaa...
		 *
		 *
		//prediction(struct Distnode *Head_distfromGrid,
		                  int numofNearestPoint,
		                  double range, double sill,
		                  int numofPoints,  --> idx_onRange
		                  int gridX, int gridY,
		                  double coord_gridX, double coord_gridY,
		                  double** distance) ptopDistance


		 */
		int N = idx_onRange+1; // we need one more column  & Row to fill 0 and 1 values (Ordinary Kriging)
		double predict=0;

		//TODO calculate distance achor node with
		int counter_idxClosestRange =0;
		while (counter_idxClosestRange < idx_onRange){
			ptopDistance[counter_idxClosestRange][N-1] = sqrt((float)(powf((float)(x_closestNodesIndex[counter_idxClosestRange]-(x_node+minimumX)),2.0)
					+powf((float)(y_closestNodesIndex[counter_idxClosestRange]-(y_node+minimumY)),2.0)));
			//cuPrintf("ptopDistance[%d][%d]:%f\n",counter_idxClosestRange,N-1,ptopDistance[counter_idxClosestRange][N-1]);
			counter_idxClosestRange++;
		}

		//		for(int it=0;it<counter_idxClosestRange+1;it++){
		//			//cuPrintf("ptopDistance: %2.2f \n",ptopDistance[it]);
		//		}

		double a[10201]; //101x101
		double *a_ptr = a;
		double rptr[101][101];//double **rptr;
		//		a = (double*) rptr;

		double b[101];
		int pivot[101];
		double sol[101];
		memset(rptr,0,sizeof(rptr));
		memset(b,0,sizeof(b));
		memset(pivot,0,sizeof(pivot));
		memset(sol,0,sizeof(sol));

		//----------- Applied Gamma Function for Ordinary Kriging
		for(i=0;i<N;i++){
			for(j=i;j<N;j++){
				if(j==N-1) {
					rptr[i][j] = -1.0;
					//cuPrintf("%f < %f kah?? \n",ptopDistance [i][j],range );
					if (ptopDistance [i][j] < range){
						b[i] = sphericalModel(range, sill, ptopDistance[i][j]);
					}
					else{
						b[i] = sill;
					}
					//					//cuPrintf("b[%d]:%f\n",i,b[i]);
				}else{
					if (ptopDistance [i][j] < range){
						rptr[i][j] = sphericalModel(range, sill, ptopDistance[i][j]);
					}
					else{
						rptr[i][j] = sill;
					}
					//					//cuPrintf("rptr[%d]:%f\n",i,rptr[i]);
					rptr[j][i] = rptr[i][j];
					rptr[N-1][j]=1.0;
				}
			}
		}
		rptr[N-1][N-1] = 0.0;
		b[N-1] = 1.0;

		//TODO change this with more efficient way
		for(i=0;i<N;i++){
			for(j=0;j<N;j++){
				//cuPrintf("rptr[%d][%d]:%f\n",i,j,rptr[i][j]);
				*a_ptr = rptr[i][j];
				a_ptr++;
			}
		}

		//		for(i=0;i<N;i++){
		//cuPrintf("b[%d]:%f\n",i,b[i]);
		//		}

		int err = Doolittle_LU_Decomposition_with_Pivoting(a, pivot,  N);
		//cuPrintf("N=%d \n",N);
		//cuPrintf("err: %d\n",err);
		//printf ("after decom\n");
		if (err < 0) {//cuPrintf("matrix is Singular\n");
			//printf(" Matrix A is singular\n");
		}
		else {
			err = Doolittle_LU_with_Pivoting_Solve(a, b, pivot, sol, N);

		}

		weightsum=0.0;
		predict = 0.0;
		//calculate predict value in grid or radius or nearest points
		for (i=0 ; i<N-1 ; i++){
			//cuPrintf("sol[%d]:%f\n",i,sol[i]);
			predict += z_closestNodesIndex[i] * sol[i];
			weightsum += sol[i];
		}

		//cuPrintf("ID[%d];weightsum: %f\n",threadID,weightsum);
		//cuPrintf("ID[%d];predict %f\n",threadID,predict);
		//		//cuPrintf("sill[%d]: %2.2f\n", threadID, sill);
		//		//cuPrintf("range[%d]: %2.2f\n", threadID, range);


	}
	int temp = threadID;
	threadID = threadID-500000*GPUIndex;
	//assign flag complete notification
	ptrIsFinished[threadID]=temp;
}

// main routine that executes on the host
int main(void)
{
	/** Read, parse  input LIDAR data & find min,max value
        and calculate range X & Y
	 **/


	clock_t start_time = clock();
	int lineNumber=0;//1186845;
	char* inputPathLIDARdata = "data/Data4_XYZ_Ground.txt";
	//	char* inputPathLIDARdata = "data/DataSample.txt";

	//Calculate number of line from input file
	static const char* filename = inputPathLIDARdata;
	FILE *filetmp = fopen ( filename, "r" );
	if ( filetmp != NULL )
	{
		char line [ 128 ];
		while ( fgets ( line, sizeof line, filetmp ) != NULL ) /* read a line */
		{
			lineNumber++; /* write the line */
		}
		fclose ( filetmp );
	}
	else
		perror ( filename );

	printf("Line Number : %d\n", lineNumber);

	int N = lineNumber;
	FILE *file=NULL;
	int i=0;

	printf("Starting Gridding ...\n");
	file=fopen(inputPathLIDARdata,"r");
	if(file==NULL){
		fprintf(stderr,"[gridding.cu] cannot open input LIDAR Data\n");
		exit(EXIT_FAILURE);
	}

	double *x=NULL;
	double *y=NULL;
	double *z=NULL;
	x = (double*)malloc(sizeof(double)*(N));
	y = (double*)malloc(sizeof(double)*(N));
	z = (double*)malloc(sizeof(double)*(N));
	for(i=0;i<N;i++){
		fscanf(file,"%lf %lf %lf", &x[i], &y[i], &z[i]);
		findMinMax(x[i], y[i]);
	}
	fclose (file);

	//round min & max value
	minY = floor(minY);
	minX = floor(minX);
	maxX = ceil(maxX);
	maxY = ceil(maxY);
	printf ("max Y %lf; min Y %lf \n" , maxY , minY);
	printf ("max X %lf; min X %lf \n" , maxX , minX);
	int gridXrange = (int) (maxX-minX );
	int gridYrange = (int) (maxY-minY );
	printf ("gridXrange %d (in meters) \n", gridXrange);
	printf ("gridYrange %d (in meters) \n", gridYrange);

	//Define variable for all CUDA devices
	double *x_devices, *x_devices2;
	double *y_devices, *y_devices2;
	double *z_devices, *z_devices2;
	double *isFinished, *isFinished2;
	double isFinished_host[500000];
	double isFinished_host2[500000];
	int device;
	double *ptrFinish, *ptrFinish2;
	ptrFinish = isFinished_host;
	ptrFinish2 = isFinished_host2;
	dim3 dimGrid(50,10);
	dim3 dimBlock(500,2);

	//Start CUDA Programming device 0
	hipSetDevice(0);
//	cudaPrintfInit();
	hipGetDevice(&device);
	cudaThreadSetLimit(hipLimitMallocHeapSize,1024*1024*1024);
	printf("dev: %d\n",device);

	hipMalloc((void**)&x_devices,sizeof(double)*(N));
	hipMalloc((void**)&y_devices,sizeof(double)*(N));
	hipMalloc((void**)&z_devices,sizeof(double)*(N));
	hipMalloc((void**)&isFinished, sizeof(double)*500000);
	hipMemset(isFinished,-1,sizeof(double)*500000);

	//	dim3 dimGrid(32,32);
	//	dim3 dimBlock(32,32);

	hipMemcpy(x_devices, x, sizeof(double)*(N), hipMemcpyHostToDevice);
	hipMemcpy(y_devices, y, sizeof(double)*(N), hipMemcpyHostToDevice);
	hipMemcpy(z_devices, z, sizeof(double)*(N), hipMemcpyHostToDevice);

	clock_t preprocessing_time = clock();

	on_core_process<<< dimGrid, dimBlock>>>(0,isFinished, x_devices, y_devices, z_devices,minX,minY, N);
	hipMemcpy(ptrFinish,isFinished,sizeof(double)*500000,hipMemcpyDeviceToHost);
	printf("CUDA err: %s \n", hipGetErrorString(hipGetLastError()));

	//Start CUDA Programming device 1
	hipSetDevice(1);
	hipGetDevice(&device);
	cudaThreadSetLimit(hipLimitMallocHeapSize,1024*1024*1024);
	printf("dev: %d\n",device);

	hipMalloc((void**)&x_devices2,sizeof(double)*(N));
	hipMalloc((void**)&y_devices2,sizeof(double)*(N));
	hipMalloc((void**)&z_devices2,sizeof(double)*(N));
	hipMalloc((void**)&isFinished2, sizeof(double)*500000);
	hipMemset(isFinished2,-1,sizeof(double)*500000);

	hipMemcpy(x_devices2, x, sizeof(double)*(N), hipMemcpyHostToDevice);
	hipMemcpy(y_devices2, y, sizeof(double)*(N), hipMemcpyHostToDevice);
	hipMemcpy(z_devices2, z, sizeof(double)*(N), hipMemcpyHostToDevice);

	on_core_process<<< dimGrid, dimBlock>>>(1,isFinished2, x_devices2, y_devices2, z_devices2,minX,minY, N);

	clock_t finished_time = clock();

	//	hipDeviceSynchronize();
	hipMemcpy(ptrFinish2,isFinished2,sizeof(double)*500000,hipMemcpyDeviceToHost);

//	printf("CUDA Synch err: %s \n", hipGetErrorString(hipDeviceSynchronize()));
	printf("CUDA last err: %s \n", hipGetErrorString(hipGetLastError()));

	clock_t postprocessing_time = clock();

	double time_preprocessing 		= ((double)(preprocessing_time-start_time))/CLOCKS_PER_SEC;
	double time_kernelProcessing 	= ((double)(finished_time-preprocessing_time))/CLOCKS_PER_SEC;
	double time_postprocessing	    = ((double)(postprocessing_time-finished_time))/CLOCKS_PER_SEC;

	int counter_done = 0;
	int counter_unexecuted = 0;
	for(int b=0;b<500000;b++){
		if((b<20))
			printf("cek (1) [%d]:%2.2f\n",b,isFinished_host[b]);
		if(isFinished_host[b]==b){
			counter_done++;
		}else{
			counter_unexecuted++;
		}
	}

	int counter_done2 = 0;
	int counter_unexecuted2 = 0;
	for(int b=0;b<500000;b++){
		if((b<20))
			printf("cek (2)[%d]:%2.2f\n",b,isFinished_host2[b]);
		if(isFinished_host2[b]==b){
			counter_done2++;
		}else{
			counter_unexecuted2++;
		}
	}


	printf("counter_done : %d (%2.2f)\% \n",counter_done,(double) counter_done/(counter_done+counter_unexecuted)*100);

	printf("total Preprocessing time %2.2f \n",time_preprocessing);
	printf("total Kernel time %2.2f \n",time_kernelProcessing);
	printf("total Postprocessing time %2.2f \n",time_postprocessing);

//	cudaPrintfDisplay();
//	cudaPrintfEnd();

	//de-allocate memory both in host and devices
	free(x);
	free(y);
	free(z);
	hipFree(x_devices);
	hipFree(y_devices);
	hipFree(z_devices);
	hipFree(x_devices2);
	hipFree(y_devices2);
	hipFree(z_devices2);
	hipFree(isFinished);
	hipFree(isFinished2);
}

void findMinMax (double x, double y){
	if (minX == 0.0 && maxX == 0.0){
		minX = x;
		minY = y;
		maxX = x;
		maxY = y;
	}

	if(minY>y)
		minY = y;
	if(minX>x)
		minX = x;
	if(maxX<x)
		maxX = x;
	if(maxY<y)
		maxY = y;
}
